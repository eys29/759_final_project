#include "hip/hip_runtime.h"
#include <iostream> 
#include <string>

#include "image.hpp"
#include "sift.hpp"
#include <chrono> 

#include <hip/hip_runtime.h>

// compile
// nvcc cudasift.cu sift.cpp image.cpp -Xcompiler -O3 -Xcompiler -O3 -std c++17 -Xcompiler -fopenmp -o cudasift --expt-relaxed-constexpr

// CUDA Kernel for distance computation
__global__ void compute_distances(std::array<uint8_t, 128> desc_a[], std::array<uint8_t, 128> desc_b[], float* distances, int num_a, int num_b, int dim) {
    int idx_a = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_b = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx_a < num_a && idx_b < num_b) {
        float dist = 0.0f;
        for (int i = 0; i < dim; i++) {
            int di = (int)desc_a[idx_a][i] - (int)desc_b[idx_b][i];
            dist += di * di;
        }
        distances[idx_a * num_b + idx_b] = sqrtf(dist);
    }
}

// Match features based on distances
__global__ void match_features(float* distances, int* matches, int num_a, int num_b, float thresh_relative, float thresh_absolute) {
    int idx_a = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_a < num_a) {
        float best_dist = 1e10f;
        float second_best_dist = 1e10f;
        int best_idx = -1;

        for (int idx_b = 0; idx_b < num_b; idx_b++) {
            float dist = distances[idx_a * num_b + idx_b];
            if (dist < best_dist) {
                second_best_dist = best_dist;
                best_dist = dist;
                best_idx = idx_b;
            } else if (dist < second_best_dist) {
                second_best_dist = dist;
            }
        }

        // Apply Lowe's ratio test
        if (best_dist < thresh_relative * second_best_dist && best_dist < thresh_absolute) {
            matches[idx_a] = best_idx;
        } else {
            matches[idx_a] = -1; // No match
        }
    }
}

int main(int argc, char *argv[])
{
    std::ios_base::sync_with_stdio(false);
    std::cin.tie(NULL);

    if (argc != 3) {
        std::cerr << "Usage: ./match_features a.jpg b.jpg (or .png)\n";
        return 0;
    }
    // keep on cpu
    Image a(argv[1]), b(argv[2]);
    a = a.channels == 1 ? a : rgb_to_grayscale(a);
    b = b.channels == 1 ? b : rgb_to_grayscale(b);

    // Start timing
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // keep on cpu
    std::vector<sift::Keypoint> kps_a = sift::find_keypoints_and_descriptors(a);
    std::vector<sift::Keypoint> kps_b = sift::find_keypoints_and_descriptors(b);
    
    int num_a = kps_a.size();
    int num_b = kps_b.size();
    int dim = 128; // Descriptor size

    // Allocate and copy descriptors to device
    std::array<uint8_t, 128> desc_a[num_a];
    std::array<uint8_t, 128> desc_b[num_b];
    hipMalloc((void **)&desc_a, num_a * sizeof(std::array<uint8_t, 128>));
    hipMalloc((void **)&desc_b, num_b * sizeof(std::array<uint8_t, 128>));

    std::array<uint8_t, 128> h_desc_a[num_a];
    std::array<uint8_t, 128> h_desc_b[num_b];
    //uint8_t* h_desc_a = (uint8_t *)malloc(num_a * dim * sizeof(uint8_t));
    //uint8_t* h_desc_b = (uint8_t *)malloc(num_b * dim * sizeof(uint8_t));

    for (int i = 0; i < num_a; i++)
        memcpy(&h_desc_a[i], (void *)&kps_a[i].descriptor, sizeof(std::array<uint8_t, 128>));

    for (int i = 0; i < num_b; i++)
        memcpy(&h_desc_b[i], (void *)&kps_b[i].descriptor, sizeof(std::array<uint8_t, 128>));

    hipMemcpy(desc_a, h_desc_a, num_a * sizeof(std::array<uint8_t, 128>), hipMemcpyHostToDevice);
    hipMemcpy(desc_b, h_desc_b, num_b * sizeof(std::array<uint8_t, 128>), hipMemcpyHostToDevice);

    // Allocate memory for distances and matches on device
    float* distances;
    int* matches;
    hipMalloc((void **)&distances, num_a * num_b * sizeof(float));
    hipMalloc((void **)&matches, num_a * sizeof(int));

    // Launch distance computation kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((num_a + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (num_b + threadsPerBlock.y - 1) / threadsPerBlock.y);

    compute_distances<<<numBlocks, threadsPerBlock>>>(desc_a, desc_b, distances, num_a, num_b, dim);
    hipDeviceSynchronize();

    // Launch matching kernel
    dim3 threadsPerMatchBlock(256);
    dim3 numMatchBlocks((num_a + threadsPerMatchBlock.x - 1) / threadsPerMatchBlock.x);

    match_features<<<numMatchBlocks, threadsPerMatchBlock>>>(distances, matches, num_a, num_b, 0.7f, 350.0f);
    hipDeviceSynchronize();

    //Timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy matches back to host
    int* h_matches = (int *)malloc(sizeof(int) * num_a);
    hipMemcpy(h_matches, matches, num_a * sizeof(int), hipMemcpyDeviceToHost);

    // Post-process matches
    std::vector<std::pair<int, int>> final_matches;
    for (int i = 0; i < num_a; i++) {
        if (h_matches[i] >= 0) {
            final_matches.push_back({i, h_matches[i]});
        }
    }
    
    
    Image result = sift::draw_matches(a, b, kps_a, kps_b, final_matches);
    result.save("result.jpg");
    
    // Output results
    std::cout << "Found " << final_matches.size() << " feature matches. Output image is saved as result.jpg\n";
    std::cout << "Execution time: " << elapsed << " milliseconds.\n";

    hipFree(desc_a);
    hipFree(desc_b);
    hipFree(distances);
    hipFree(matches);
    //free(h_desc_a);
    //free(h_desc_b);
    free(h_matches);


    return 0;
}
